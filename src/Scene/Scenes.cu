#include "hip/hip_runtime.h"
#include <Scenes.cuh>
#include <Materials.cuh>
#include <Objects.cuh>
#include <Textures.cuh>
#include <Transformations.cuh>
#include <Light.cuh>
#include <Pdfs.cuh>

namespace scene
{
    SCENE(cornell_box)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        auto red = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.65, .05, .05));
        auto white = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.73, .73, .73));
        auto green = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.12, .45, .15));
        auto light = SharedPointer::makeShared<DiffuseLight>(SharedPointer::makeShared<SolidColor>(15., 15., 15.));
        auto aluminum = SharedPointer::makeShared<Metal>(SharedPointer::makeShared<SolidColor>(.8, .85, .88), 0.);

        objects->add(SharedPointer::makeShared<FlipFace>(SharedPointer::makeShared<AARect<utils::Axis::X>>(0., 555., 0., 555., 555., green)));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::X>>(0., 555., 0., 555., 0., red));
        objects->add(SharedPointer::makeShared<FlipFace>(SharedPointer::makeShared<AARect<utils::Axis::Y>>(213., 343., 227., 332., 554., light)));
        objects->add(SharedPointer::makeShared<FlipFace>(SharedPointer::makeShared<AARect<utils::Axis::Y>>(0., 555., 0., 555., 0., white)));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(0., 555., 0., 555., 555., white));
        objects->add(SharedPointer::makeShared<FlipFace>(SharedPointer::makeShared<AARect<utils::Axis::Z>>(0., 555., 0., 555., 555., white)));

        // SharedPointer<Hittable> box1 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 330., 165.}, SharedPointer::makeShared<Metal>(SharedPointer::makeShared<SolidColor>(.8, .85, .88), 0.));
        SharedPointer<Hittable> box1 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 330., 165.}, aluminum);

        box1 = SharedPointer::makeShared<AARotate<utils::Axis::Y>>(box1, 15.);
        box1 = SharedPointer::makeShared<Translate>(box1, Vec3{300., 0., 295.});

        SharedPointer<Hittable> box3 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 400., 165.}, white);

        box3 = SharedPointer::makeShared<AARotate<utils::Axis::Y>>(box3, -20.);
        box3 = SharedPointer::makeShared<Translate>(box3, Vec3{90., 0., 295.});

        SharedPointer<Hittable> box2 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 165., 165.}, white);

        box2 = SharedPointer::makeShared<AARotate<utils::Axis::Y>>(box2, -18.);
        box2 = SharedPointer::makeShared<Translate>(box2, Vec3{130., 0., 65.});

        objects->add(box1);
        objects->add(box3);
        // objects->add(SharedPointer::makeShared<Sphere>(Vec3{190., 390., 190.}, 90., red));
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{190., 90., 190.}, 90., SharedPointer::makeShared<Dielectric>(1.5)));
        // objects->add(box2);

        sampleObjects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(213., 343., 227., 332., 554., 
                SharedPointer::makeShared<Material>(nullptr)));
        sampleObjects->add(SharedPointer::makeShared<Sphere>(Vec3{190., 90., 190.}, 90., SharedPointer::makeShared<Material>(nullptr)));
    }

    SCENE(perlin_spheres)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        auto pertext = SharedPointer::makeShared<TurbulentTexture>();
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., -1000., 0.}, 1000., SharedPointer::makeShared<LambertianDiffuse>(pertext)));
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., 2., 0.}, 2., SharedPointer::makeShared<LambertianDiffuse>(pertext)));
    }

    SCENE(image_texture)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        auto imgtext = SharedPointer::makeShared<ImageTexture>("world.jpg");
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., 0., 0.}, 2., SharedPointer::makeShared<LambertianDiffuse>(imgtext)));
    }

    SCENE(light_scene)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        auto difflight = SharedPointer::makeShared<DiffuseLight>(SharedPointer::makeShared<SolidColor>(4., 4., 4.));
        auto mat = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(1., 0., 0.));

        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., -1000., 0.}, 1000., mat));
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., 2., 0.}, 2., mat));

        objects->add(SharedPointer::makeShared<Sphere>(Vec3{0., 7., 0.}, 2., difflight));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Z>>(3., 5., 1., 3., -2., difflight));
    }

    SCENE(volume_cornell_box)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        auto red = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.65, .05, .05));
        auto white = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.73, .73, .73));
        auto green = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.12, .45, .15));
        auto light = SharedPointer::makeShared<DiffuseLight>(SharedPointer::makeShared<SolidColor>(15., 15., 15.));

        objects->add(SharedPointer::makeShared<AARect<utils::Axis::X>>(0., 555., 0., 555., 555., green));

        objects->add(SharedPointer::makeShared<AARect<utils::Axis::X>>(0., 555., 0., 555., 0., red));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(113., 443., 127., 432., 554., light));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(0., 555., 0., 555., 0., white));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(0., 555., 0., 555., 555., white));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Z>>(0., 555., 0., 555., 555., white));

        SharedPointer<Hittable> box1 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 330., 165.}, white);

        box1 = SharedPointer::makeShared<AARotate<utils::Axis::Y>>(box1, 15.);
        box1 = SharedPointer::makeShared<Translate>(box1, Vec3{265., 0., 295.});

        SharedPointer<Hittable> box2 = SharedPointer::makeShared<Box>(Vec3{0., 0., 0.}, Vec3{165., 165., 165.}, white);

        box2 = SharedPointer::makeShared<AARotate<utils::Axis::Y>>(box2, -18.);
        box2 = SharedPointer::makeShared<Translate>(box2, Vec3{130., 0., 65.});

        objects->add(SharedPointer::makeShared<ConstantVolume>(box1, SharedPointer::makeShared<SolidColor>(0., 0., 0.), .01));
        objects->add(SharedPointer::makeShared<ConstantVolume>(box2, SharedPointer::makeShared<SolidColor>(1., 1., 1.), .01));
    }

    SCENE(summary)
    {
        const double fieldOfView = 40.;
        const double apertureRadius = 0.;
        const double distanceToFocus = 10.;
        const Vec3 lookFrom = Vec3{278., 278., -800.};
        const Vec3 lookAt = Vec3{278., 278., 0.};
        const double t0 = 0.;
        const double t1 = 1.;

        camera = SharedPointer::makeShared<Camera>(aspectR, fieldOfView, apertureRadius, distanceToFocus, lookFrom, lookAt, t0, t1);

        SharedPointer<HittableList> boxes1 = SharedPointer::makeShared<HittableList>();

        auto ground = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.48, .83, .53));

        const int boxes_per_side = 20;
        for (int i = 0; i < boxes_per_side; ++i)
        {
            for (int j = 0; j < boxes_per_side; ++j)
            {
                auto w = 100.;
                auto x0 = -1000. + i * w;
                auto z0 = -1000. + j * w;
                auto y0 = 0.;
                auto x1 = x0 + w;
                auto y1 = utils::random_double(1., 101.);
                auto z1 = z0 + w;

                boxes1->add(SharedPointer::makeShared<Box>(Vec3{x0, y0, z0}, Vec3{x1, y1, z1}, ground));
            }
        }

        objects = SharedPointer::makeShared<HittableList>();
        sampleObjects = SharedPointer::makeShared<HittableList>();

        objects->add(SharedPointer::makeShared<BVHNode>(*boxes1.get(), 0., 1.));
        auto light = SharedPointer::makeShared<DiffuseLight>(SharedPointer::makeShared<SolidColor>(7., 7., 7.));
        objects->add(SharedPointer::makeShared<AARect<utils::Axis::Y>>(123., 423., 147., 412., 554., light));

        auto center1 = Vec3{400., 400., 200.};
        auto center2 = center1 + Vec3{30., 0., 0.};
        auto moving_sphere_material =
            SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.7, .3, .1));
        objects->add(SharedPointer::makeShared<Sphere>(center1, center2, 50., moving_sphere_material));

        objects->add(SharedPointer::makeShared<Sphere>(Vec3{260., 150., 45.}, 50., SharedPointer::makeShared<Dielectric>(1.5)));
        objects->add(SharedPointer::makeShared<Sphere>(
            Vec3{0., 150., 145.}, 50., SharedPointer::makeShared<Metal>(SharedPointer::makeShared<SolidColor>(Vec3{.8, .8, .9}), 10.)));

        auto boundary = SharedPointer::makeShared<Sphere>(Vec3{360., 150., 145.}, 70., SharedPointer::makeShared<Dielectric>(1.5));
        objects->add(boundary);
        objects->add(SharedPointer::makeShared<ConstantVolume>(
            boundary, SharedPointer::makeShared<SolidColor>(.2, .4, .9), .2));
        boundary = SharedPointer::makeShared<Sphere>(Vec3{0., 0., 0.}, 5000., SharedPointer::makeShared<Dielectric>(1.5));
        objects->add(SharedPointer::makeShared<ConstantVolume>(
            boundary, SharedPointer::makeShared<SolidColor>(1., 1., 1.), .0001));

        auto emat = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<ImageTexture>("world.jpg"));
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{400., 200., 400.}, 100., emat));
        auto pertext = SharedPointer::makeShared<PerlinNoiseTexture>(.1);
        objects->add(SharedPointer::makeShared<Sphere>(Vec3{220, 280., 300.}, 80., SharedPointer::makeShared<LambertianDiffuse>(pertext)));

        SharedPointer<HittableList> boxes2 = SharedPointer::makeShared<HittableList>();
        
        auto white = SharedPointer::makeShared<LambertianDiffuse>(SharedPointer::makeShared<SolidColor>(.73, .73, .73));
        int ns = 1000;
        for (int j = 0; j < ns; ++j)
        {
            boxes2->add(SharedPointer::makeShared<Sphere>(Vec3::randomVector(Vec3{}, Vec3{165., 165., 165.}), 10., white));
        }

        objects->add(SharedPointer::makeShared<Translate>(
            SharedPointer::makeShared<AARotate<utils::Axis::Y>>(
                SharedPointer::makeShared<BVHNode>(*boxes2.get(), 0., 1.), 15.),
            Vec3{-100., 270., 395.}));
    }

} // namespace scene
