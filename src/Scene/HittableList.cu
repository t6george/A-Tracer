#include <cassert>
#include <cstring>

#include <HittableList.cuh>
#include <AABB.cuh>

#if GPU == 1
HittableList::HittableNode::HittableNode(const SharedPointer<Hittable>& data) : next{nullptr}, data{data} {}

HittableList::HittableNode::~HittableNode()
{
    HittableNode* tmp = next;
    hipFree(this);
    hipFree(next);
}

HittableList::HittableLinkedList::HittableLinkedList() : head{nullptr}, tail{nullptr}, size{0} {}

HittableList::HittableLinkedList::~HittableLinkedList()
{
    clear();
}

void HittableList::HittableLinkedList::emplace_back(const SharedPointer<Hittable>& data)
{
    HittableNode* newNode = nullptr;
    HittableNode node(data);

    hipMallocManaged((void**) &newNode, sizeof(HittableNode));
    memmove((void*) newNode, (void*) &node, sizeof(HittableNode));
    
    if (head)
    {
        tail->next = newNode;
	tail = tail->next;
    }
    else
    {
        head = tail = newNode;
    }

    ++size;
}

void HittableList::HittableLinkedList::clear()
{
    hipFree(head);
    head = tail = nullptr;
    size = 0;
}

bool empty() const
{
    return size == 0;
}

SharedPointer<Hittable> HittableList::HittableLinkedList::at(unsigned i) const
{
    SharedPointer<Hittable> hittable;
    HittableNode itr = head;

    for (unsigned j = 0; j < i && itr; ++j)
    {
        itr = itr->next;
    }
    
    if (itr)
    {
        hittable = itr->data;
    }

    return hittable;
}

#endif

DEV Hittable::HitType HittableList::getCollisionData(const Ray &ray, HitRecord &record,
                             double tMin, double tMax, bool flip) const
{
    Hittable::HitRecord tmpRecord;
    Hittable::HitType collisionType = Hittable::HitType::NO_HIT, tmpCollisionType;

    for (const auto &obj : hittables)
    {
        if (static_cast<bool>(tmpCollisionType = obj.get()->getCollisionData(ray, tmpRecord, tMin, tMax, flip)))
        {
            collisionType = tmpCollisionType;
            record = tmpRecord;
            tMax = record.t;
        }
    }

    return collisionType;
}

DEV bool HittableList::getBoundingBox(double time0, double time1, AABB &box) const
{
    bool firstBox = true;
    AABB tmp, outputBox;

    for (const auto &obj : hittables)
    {
        if (!obj->getBoundingBox(time0, time1, tmp))
            return false;
        outputBox = firstBox ? tmp : AABB::combineAABBs(outputBox, tmp);
        firstBox = false;
    }

    return !hittables.empty();
}

HOST void HittableList::add(SharedPointer<Hittable> hittable)
{
    hittables.emplace_back(hittable);
}

HOST void HittableList::clear()
{
    hittables.clear(); 
}

DEV Vec3 HittableList::genRandomVector(const Vec3& origin) const
{
    return hittables.at(utils::random_int(0, hittables.size()))->genRandomVector(origin);
}

DEV double HittableList::eval(const Vec3& origin, const Vec3& v, bool flip) const
{
    assert(hittables.size() > 0);
    double weight = 1. / hittables.size();
    double sum = 0.;

    for (const auto& hittable : hittables)
    {
        sum += weight * hittable->eval(origin, v, flip);
    }

    return sum;
}
